#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include ".\lodepng.h"
#include <algorithm> 
#include<time.h>


using namespace std;

__device__ int max(int a, int b, int c, int d) {
	int max = a;
	if (b > max) max = b;
	if (c > max) max = c;
	if (d > max) max = d;
	return max;
}

__global__ void process(unsigned char* input_image, unsigned char* output_image, unsigned width, unsigned height, int num_threads)
{
	int start;
	int end;
	// Number of pixels to process
	int total_size = width * height;
	// Number of pixels per thread
    // 994 x 998 = 992 012
	int thread_size = total_size / num_threads;
	int blocks_per_thread = thread_size / 4;
    int blocks_per_row = width/2; 

	start = blocks_per_thread * threadIdx.x;
	end = start + blocks_per_thread;

    if (num_threads > total_size / 4) {
        blocks_per_thread = 1;
    }
	 
    // process image
	// split image into N 2x2 blocks
	// each thread processes N/numThreads blocks 
	// first square: tid * 8 (tid = 0) 
	// below first: tid*8 + width * 8
    // i is block number
    for (int i = start; i< end; i++){
        int row = 2*i/blocks_per_row;
        unsigned char* one = input_image + i % blocks_per_row * 4 * 2 + row * width*4;
        unsigned char* two = one + 4;
        unsigned char* three = one+ width*4;
        unsigned char* four = two + width*4;

		int maxR = max( *one, *two, *three, *four );
		int maxG = max( *(one+1), *(two+1), *(three+1), *(four+1));
		int maxB = max( *(one + 2), *(two + 2), *(three + 2), *(four + 2) );
		int maxA = max( *(one + 3), *(two + 3), *(three + 3), *(four + 3) );

		output_image[4 * i] = maxR;
		output_image[4 * i + 1] = maxG;
		output_image[4 * i + 2] = maxB;
		output_image[4 * i + 3] = maxA;
    }
}

int main(int argc, char* argv[])
{
    char* input_filename = argv[1];
    char* output_filename = argv[2];
    //double time_spent = 0.0;
    int thread_nums = atoi(argv[3]);

    unsigned error;
    unsigned char* image, * new_image;
    unsigned char* d_image; 
    unsigned width, height;

    error = lodepng_decode32_file(&image, &width, &height, input_filename);
    if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
    
    //  allocated memory in the device for the input image
    // we dont need it again the the host so just do hipMalloc
    size_t imageSize = (size_t) width * height * 4 * sizeof(unsigned char);
    hipMalloc((void** ) & d_image, imageSize);
    hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice);
    // allocate shared memory for the new image because we want it in host
    hipMallocManaged(&new_image, imageSize/4);

    double time_spent = 0.0;
    clock_t begin = clock();

    process << <1, thread_nums >> > (d_image, new_image, width, height, thread_nums);

    //process<<<1, thread_nums >>>(d_image, new_image, width, height, thread_nums);
    //process(d_image, new_image, width, height, num_threads);

    hipDeviceSynchronize();

    lodepng_encode32_file(output_filename, new_image, width/2, height/2);

    clock_t end = clock();
    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Number of threads: %d    Run time %f   \n", thread_nums, time_spent);

    hipFree(d_image); hipFree(new_image);
    return 0;
}
